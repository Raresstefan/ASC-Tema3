#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"
#define LOAD_FACTOR_MIN 0.5f
#define LOAD_FACTOR_MAX 1.0f

using namespace std;

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

hipError_t getNumBlocksThreads(int *numBlocks, int *numThreads, int nr) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    *numThreads = deviceProp.maxThreadsPerBlock;
    *numBlocks = nr / (*numThreads);
    if (*numBlocks * (*numThreads) != nr) {
        (*numBlocks)++;
    }
    return hipSuccess;
}

// Calculates hash for a key using the algorithm described here:
//https://burtleburtle.net/bob/hash/integer.html
static __device__ size_t calculateHash(int key) {
	size_t keyHash = (size_t) key;
    keyHash -= (keyHash << 6);
    keyHash ^= (keyHash >> 17);
    keyHash -= (keyHash << 9);
    keyHash ^= (keyHash << 4);
    keyHash -= (keyHash << 3);
    keyHash ^= (keyHash << 10);
    keyHash ^= (keyHash >> 15);
    return keyHash;
}

static __global__ void insert_entry(HashElement *hashTable, int *keys,
    int *values, int *nrUpdates, int maxElements)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > maxElements) {
        return;
    }
    size_t computedHash = calculateHash(keys[idx]) % maxElements;
	int currentKey = atomicCAS(&hashTable[computedHash].key, 0, keys[idx]);
    while (currentKey != 0 && keys[idx] != currentKey) {
        computedHash = (computedHash + 1) % maxElements;
		currentKey = atomicCAS(&hashTable[computedHash].key, 0, keys[idx]);
    }
	if (currentKey == keys[idx]) {
		atomicAdd(nrUpdates, 1);
	}
	hashTable[computedHash].value = values[idx];
}

static __global__ void reshape_table(HashElement *oldTable, HashElement *newTable,
	int oldSize, int newSize)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > oldSize || oldTable[idx].key == 0) {
		return;
	}
	size_t computedHash = calculateHash(oldTable[idx].key) % newSize;
	bool readded = false;
	while (!readded) {
		if (atomicCAS(&newTable[computedHash].key, 0, oldTable[idx].key) == 0) {
			newTable[computedHash].value = oldTable[idx].value;
			readded = true;
		}
		computedHash = (computedHash + 1) % newSize;
	}
}

static __global__ void get_entry(HashElement *hashTable, int *keys,
	int *values, int maxElements, int nrKeys)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > nrKeys) {
		return;
	}
	size_t computedHash = calculateHash(keys[idx]) % maxElements;
	while (hashTable[computedHash].key != keys[idx]) {
		computedHash = (computedHash + 1) % maxElements;
	}
	values[idx] = hashTable[computedHash].value;
}

/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) {
    maxElements = size;
    nrElements = 0;
    hipMallocManaged(&hashTable, maxElements * sizeof(*hashTable));
    hipMemset(hashTable, 0, maxElements * sizeof(*hashTable));
}

float GpuHashTable::loadFactor() {
    return nrElements / float(maxElements);
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
    hipFree(hashTable);
}

/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	HashElement *reshaped;
	int nrBlocks, nrThreads;
	hipMallocManaged(&reshaped, numBucketsReshape * sizeof(*reshaped));
	hipMemset(reshaped, 0, numBucketsReshape * sizeof(*reshaped));
	getNumBlocksThreads(&nrBlocks, &nrThreads, maxElements);
	reshape_table<<<nrBlocks, nrThreads>>>(hashTable, reshaped, maxElements, numBucketsReshape);
	hipDeviceSynchronize();
	hipFree(hashTable);
	hashTable = reshaped;
	maxElements = numBucketsReshape;
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
    int *keysCopy;
    int *valuesCopy;
    int *updates;
    int nrBlocks, nrThreads;
    hipMallocManaged(&keysCopy, numKeys * sizeof(int));
    hipMemcpy(keysCopy, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
    hipMallocManaged(&valuesCopy, numKeys * sizeof(int));
    hipMemcpy(valuesCopy, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
    hipMallocManaged(&updates, sizeof(int));
    if ((nrElements + numKeys) / float(maxElements) >= LOAD_FACTOR_MAX) {
        reshape((nrElements + numKeys) / LOAD_FACTOR_MIN);
    }
    getNumBlocksThreads(&nrBlocks, &nrThreads, numKeys);
	// insert part
    insert_entry<<<nrBlocks, nrThreads>>>(hashTable, keysCopy, valuesCopy, updates, maxElements);
    hipDeviceSynchronize();
    nrElements += numKeys - *updates;
	hipFree(keysCopy);
    hipFree(valuesCopy);
    return true;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
    int *values;
	int *keysCopy;
	int nrBlocks, nrThreads;
	hipMallocManaged(&keysCopy, numKeys * sizeof(int));
	hipMemcpy(keysCopy, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMallocManaged(&values, numKeys * sizeof(int));
	getNumBlocksThreads(&nrBlocks, &nrThreads, numKeys);
	// get part
	get_entry<<<nrBlocks, nrThreads>>>(hashTable, keysCopy, values, maxElements, numKeys);
	hipDeviceSynchronize();
	hipFree(keysCopy);
	return values;
}

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()